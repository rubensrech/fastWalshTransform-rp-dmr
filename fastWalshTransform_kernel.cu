/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#ifndef FWT_KERNEL_CU
#define FWT_KERNEL_CU

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#define CHECK_CUDA_ERROR(ans) { checkCudaError((ans), __FILE__, __LINE__); }
inline void checkCudaError(hipError_t code, const char *file, int line, bool abort=false) {
    if (code != hipSuccess) {
        printf("CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

///////////////////////////////////////////////////////////////////////////////
// Elementary(for vectors less than elementary size) in-shared memory 
// combined radix-2 + radix-4 Fast Walsh Transform 
///////////////////////////////////////////////////////////////////////////////
#define ELEMENTARY_LOG2SIZE 11

__global__ void fwtBatch1Kernel(double *d_Output, double *d_Input, int log2N) {
    const int N = 1 << log2N;
    int stride = N;
    const int base = blockIdx.x << log2N;

    // (2 ** 11) * 4 bytes == 8KB -- maximum s_data[] size for G80
    extern __shared__ double s_data[];
    double *d_Src = d_Input  + base;
    double *d_Dst = d_Output + base;

    for(int pos = threadIdx.x; pos < N; pos += blockDim.x)
        s_data[pos] = d_Src[pos];

    //Do single radix-2 stage if for odd power
    if(log2N & 1){
        __syncthreads();
        stride >>= 1;
        for(int pos = threadIdx.x; pos < N / 2; pos += blockDim.x){
            int lo = pos & (stride - 1);
            int i0 = ((pos - lo) << 1) + lo;
            int i1 = i0 + stride;

            double t0 = s_data[i0];
            double t1 = s_data[i1];
            s_data[i0] = t0 + t1;
            s_data[i1] = t0 - t1;
        }
    }

    //Main radix4 stages
    stride >>= 2;
    int pos = threadIdx.x;
    for(; stride >= 1; stride >>= 2){
        __syncthreads();
        int lo = pos & (stride - 1);
        int i0 = ((pos - lo) << 2) + lo;
        int i1 = i0 + stride;
        int i2 = i1 + stride;
        int i3 = i2 + stride;

        double d0 = s_data[i0];
        double d1 = s_data[i1];
        double d2 = s_data[i2];
        double d3 = s_data[i3];

        double t;
        t = d0; d0         = d0 + d2; d2         = t - d2;
        t = d1; d1         = d1 + d3; d3         = t - d3;
        t = d0; s_data[i0] = d0 + d1; s_data[i1] = t - d1;
        t = d2; s_data[i2] = d2 + d3; s_data[i3] = t - d3;
    }

    __syncthreads();
    for(int pos = threadIdx.x; pos < N; pos += blockDim.x)
        d_Dst[pos] = s_data[pos];
}

__global__ void fwtBatch1Kernel(float *d_Output, float *d_Input, int log2N) {
    const int N = 1 << log2N;
    int stride = N;
    const int base = blockIdx.x << log2N;

    // (2 ** 11) * 4 bytes == 8KB -- maximum s_data[] size for G80
    extern __shared__ float s_data_rp[];
    float *d_Src = d_Input  + base;
    float *d_Dst = d_Output + base;

    for(int pos = threadIdx.x; pos < N; pos += blockDim.x)
        s_data_rp[pos] = d_Src[pos];

    //Do single radix-2 stage if for odd power
    if(log2N & 1){
        __syncthreads();
        stride >>= 1;
        for(int pos = threadIdx.x; pos < N / 2; pos += blockDim.x){
            int lo = pos & (stride - 1);
            int i0 = ((pos - lo) << 1) + lo;
            int i1 = i0 + stride;

            float t0 = s_data_rp[i0];
            float t1 = s_data_rp[i1];
            s_data_rp[i0] = t0 + t1;
            s_data_rp[i1] = t0 - t1;
        }
    }

    //Main radix4 stages
    stride >>= 2;
    int pos = threadIdx.x;
    for(; stride >= 1; stride >>= 2){
        __syncthreads();
        int lo = pos & (stride - 1);
        int i0 = ((pos - lo) << 2) + lo;
        int i1 = i0 + stride;
        int i2 = i1 + stride;
        int i3 = i2 + stride;

        float d0 = s_data_rp[i0];
        float d1 = s_data_rp[i1];
        float d2 = s_data_rp[i2];
        float d3 = s_data_rp[i3];

        float t;
        t = d0; d0         = d0 + d2; d2         = t - d2;
        t = d1; d1         = d1 + d3; d3         = t - d3;
        t = d0; s_data_rp[i0] = d0 + d1; s_data_rp[i1] = t - d1;
        t = d2; s_data_rp[i2] = d2 + d3; s_data_rp[i3] = t - d3;
    }

    __syncthreads();
    for(int pos = threadIdx.x; pos < N; pos += blockDim.x)
        d_Dst[pos] = s_data_rp[pos];
}

////////////////////////////////////////////////////////////////////////////////
// Single in-global memory radix-4 Fast Walsh Transform pass
// (for strides exceeding elementary vector size)
////////////////////////////////////////////////////////////////////////////////
__global__ void fwtBatch2Kernel(double *d_Output, double *d_Input, int stride) {
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int   N = blockDim.x *  gridDim.x * 4;

    double *d_Src = d_Input  + blockIdx.y * N;
    double *d_Dst = d_Output + blockIdx.y * N;

    int lo = pos & (stride - 1);
    int i0 = ((pos - lo) << 2) + lo;
    int i1 = i0 + stride;
    int i2 = i1 + stride;
    int i3 = i2 + stride;

    double d0 = d_Src[i0];
    double d1 = d_Src[i1];
    double d2 = d_Src[i2];
    double d3 = d_Src[i3];

    double t;
    t = d0; d0        = d0 + d2; d2        = t - d2;
    t = d1; d1        = d1 + d3; d3        = t - d3;
    t = d0; d_Dst[i0] = d0 + d1; d_Dst[i1] = t - d1;
    t = d2; d_Dst[i2] = d2 + d3; d_Dst[i3] = t - d3;
}

__global__ void fwtBatch2Kernel(float *d_Output, float *d_Input, int stride) {
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int   N = blockDim.x *  gridDim.x * 4;

    float *d_Src = d_Input  + blockIdx.y * N;
    float *d_Dst = d_Output + blockIdx.y * N;

    int lo = pos & (stride - 1);
    int i0 = ((pos - lo) << 2) + lo;
    int i1 = i0 + stride;
    int i2 = i1 + stride;
    int i3 = i2 + stride;

    float d0 = d_Src[i0];
    float d1 = d_Src[i1];
    float d2 = d_Src[i2];
    float d3 = d_Src[i3];

    float t;
    t = d0; d0        = d0 + d2; d2        = t - d2;
    t = d1; d1        = d1 + d3; d3        = t - d3;
    t = d0; d_Dst[i0] = d0 + d1; d_Dst[i1] = t - d1;
    t = d2; d_Dst[i2] = d2 + d3; d_Dst[i3] = t - d3;
}


////////////////////////////////////////////////////////////////////////////////
// Put everything together: batched Fast Walsh Transform CPU front-end
////////////////////////////////////////////////////////////////////////////////
void fwtBatchGPU(double *d_Data, int M, int log2N) {
    int N = 1 << log2N;
    dim3 grid((1 << log2N) / 1024, M, 1);
    for(; log2N > ELEMENTARY_LOG2SIZE; log2N -= 2, N >>= 2, M <<= 2){
        fwtBatch2Kernel<<<grid, 256>>>(d_Data, d_Data, N / 4);
        CHECK_CUDA_ERROR(hipPeekAtLastError());
    }

    fwtBatch1Kernel<<<M, N / 4, N * sizeof(double)>>>(d_Data, d_Data, log2N);
    CHECK_CUDA_ERROR(hipPeekAtLastError());
}

void fwtBatchGPU(float *d_Data, int M, int log2N) {
    int N = 1 << log2N;
    dim3 grid((1 << log2N) / 1024, M, 1);
    for(; log2N > ELEMENTARY_LOG2SIZE; log2N -= 2, N >>= 2, M <<= 2){
        fwtBatch2Kernel<<<grid, 256>>>(d_Data, d_Data, N / 4);
        CHECK_CUDA_ERROR(hipPeekAtLastError());
    }

    fwtBatch1Kernel<<<M, N / 4, N * sizeof(float)>>>(d_Data, d_Data, log2N);
    CHECK_CUDA_ERROR(hipPeekAtLastError());
}

////////////////////////////////////////////////////////////////////////////////
// Modulate two arrays
////////////////////////////////////////////////////////////////////////////////
__global__ void modulateKernel(double *d_A, double *d_B, int N){
    int        tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    double     rcpN = 1.0f / (double)N;

    for(int pos = tid; pos < N; pos += numThreads)
        d_A[pos] *= d_B[pos] * rcpN;
}

__global__ void modulateKernel(float *d_A, float *d_B, int N){
    int        tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    float     rcpN = 1.0f / (float)N;

    for(int pos = tid; pos < N; pos += numThreads)
        d_A[pos] *= d_B[pos] * rcpN;
}

//Interface to modulateKernel()
void modulateGPU(double *d_A, double *d_B, int N) {
    modulateKernel<<<128, 256>>>(d_A, d_B, N);
}

void modulateGPU(float *d_A, float *d_B, int N) {
    modulateKernel<<<128, 256>>>(d_A, d_B, N);
}

////////////////////////////////////////////////////////////////////////////////
// Calculate output relative error
////////////////////////////////////////////////////////////////////////////////
__forceinline__  __device__ float relative_error(double rhs, float lhs) {
	return __fdividef(lhs, float(rhs));
}

__global__ void relative_error_kernel(double *output, float *output_rp, float *err_output, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
        err_output[tid] = relative_error(output[tid], output_rp[tid]);
}

void relative_error_gpu(double *output, float *output_rp, float *err_output, int N) {
    int blockSize = 32;
    int gridDim = (N + blockSize - 1) / blockSize;
    relative_error_kernel<<<gridDim, blockSize>>>(output, output_rp, err_output, N);
}

#endif
